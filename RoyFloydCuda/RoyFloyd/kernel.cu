#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>



__global__ void Compute_Path(int *Md, const int Width, const int k)
{
	//2 Thread ID
	int ROW = blockIdx.x;
	int COL = threadIdx.x;


	if (Md[ROW * Width + COL] > Md[ROW * Width + k] + Md[k * Width + COL])
		Md[ROW * Width + COL] = Md[ROW * Width + k] + Md[k * Width + COL];
}

void addWithCuda(int *c, int size);
int main()
{
	int d;
	int a[36] = {	0, 2, 5, 999, 999, 999, 
					999, 0, 7, 1, 999, 8,
					999, 999, 0, 4, 999, 999,
					999, 999, 999, 0, 3, 999,
					999, 999, 2, 999, 0, 3,
					999, 5, 999, 2, 4, 0
				};

	// Add vectors in parallel.
	addWithCuda(a, 6);


	for (int i = 0; i < 36; i++)
		printf("%d ", a[i]);
		
		

		

	
	scanf("%d",&d);

	return 0;
} 

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int *c, int Width)
{
	int *dev_c;
	int size = Width*Width * sizeof(int);
	// Choose which GPU to run on, change this on a multi-GPU system.
	
	

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_c, size * sizeof(int));
	

	

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
	

	// Launch a kernel on the GPU with one thread for each element.
	for (int k = 0; k < 6; k++)
		Compute_Path << <Width, Width >> >(dev_c, Width,k);

	

	// Copy output vector from GPU buffer to host memory.
	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);


	hipFree(dev_c);
}
